#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "org_pepsoft_worldpainter_exporting_gpuacceleration_ResourceNoiseGenerationRequest.h"
#include <stdlib.h>
#include "NoiseGeneration.h"
#include <windows.h>
#include <jni.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "JavaClassWrappers/GPUNoiseRequest.h"
#include "Inputs/NoiseInput.h"
#include "JavaClassWrappers/JavaRandom.h"

#include <chrono>
#include <stdio.h>
#include <ctime>

#define TILE_SIZE 128
#define REGION_SIZE 512
#define CHUNK_SIZE 16
#define MAX_HEIGHT 128
#define MIN_HEIGHT -64
#define X_ARRAY_SIZE 512
#define Y_ARRAY_SIZE 512
#define DEBUGGING false
#define height 32

hipError_t noiseWithCuda(float* chances, NoiseInput& noiseInput, int*& output, int& outputSize, int totalHeight, GPUMemoryBlock gpuMemoryBlock);
void getRegionArray(float* regionArrayX, float* regionArrayY, float* regionArrayZ, ResourceNoiseGenerationRequest resourceNoiseGenerationRequest);
void getPArray(int* p, JNIEnv* env, jlong seed);
void swap(int* array, int index1, int index2);
void freeCudaMemory(int* dev_p, float* dev_regionArrayX, float* dev_regionArrayY, float* dev_regionArrayZ, bool* dev_output, int* dev_compactedOutput);


__global__ void generateNoise(int* p, float* chances, float* regionArrayX, float* regionArrayY, float* regionArrayZ, bool *output, int totalHeight)
{
    float outputNoise = getPerlinNoiseAt(regionArrayX[blockIdx.x], regionArrayY[blockIdx.y], regionArrayZ[threadIdx.x], p);

    bool shouldSetMaterial = outputNoise >= chances[8];

    output[blockIdx.x + (blockIdx.y * X_ARRAY_SIZE) + (threadIdx.x * X_ARRAY_SIZE * X_ARRAY_SIZE)] = shouldSetMaterial;

}

struct is_true {
    __host__ __device__
        bool operator() (const bool success) {
        return success;
    }
};



// Helper function for using CUDA to add vectors in parallel.
hipError_t noiseWithCuda(float* chances, NoiseInput* noiseInput, int*& output, int& outputSize, int totalHeight, GPUMemoryBlock gpuMemoryBlock)
{
    std::clock_t c_start = std::clock();

    int* dev_p;
    float* dev_regionArrayX;
    float* dev_regionArrayY;
    float* dev_regionArrayZ;
    bool* dev_output;
    int* dev_CompactedOutput;
    float* dev_chances;

    const int MAX_OUTPUT_SIZE = X_ARRAY_SIZE * Y_ARRAY_SIZE * height;
    const int CURRENT_OUTPUT_SIZE = X_ARRAY_SIZE * Y_ARRAY_SIZE * totalHeight;


    hipError_t cudaStatus;
    auto t_start = std::chrono::high_resolution_clock::now();
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    if (gpuMemoryBlock.getpGPUPointer() == 0){
        cudaStatus = hipMalloc((void**)&dev_p, 512 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_p = (int*)gpuMemoryBlock.getpGPUPointer();
    }

    cudaStatus = hipMalloc((void**)&dev_chances, 16 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    if (gpuMemoryBlock.getxGPUPointer() == 0) {
    cudaStatus = hipMalloc((void**)&dev_regionArrayX, X_ARRAY_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        fprintf(stderr, hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }
    }
    else { //reuse
        dev_regionArrayX = (float*)gpuMemoryBlock.getxGPUPointer();
    }

    if (gpuMemoryBlock.getyGPUPointer() == 0) {
        cudaStatus = hipMalloc((void**)&dev_regionArrayY, Y_ARRAY_SIZE * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_regionArrayY = (float*)gpuMemoryBlock.getyGPUPointer();
    }

    if (gpuMemoryBlock.getzGPUPointer() == 0) {
        cudaStatus = hipMalloc((void**)&dev_regionArrayZ, height * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_regionArrayZ = (float*)gpuMemoryBlock.getzGPUPointer();
    }

    if (gpuMemoryBlock.getOutputGPUPointer() == 0)
    {
        cudaStatus = hipMalloc((void**)&dev_output,sizeof(bool)* MAX_OUTPUT_SIZE); //Varies by height but roughly 25MB
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_output = (bool*)gpuMemoryBlock.getOutputGPUPointer();
    }
    if (gpuMemoryBlock.getCompactedOutputGPUPointer() == 0) {
        cudaStatus = hipMalloc((void**)&dev_CompactedOutput, (MAX_OUTPUT_SIZE) * sizeof(int)); // Varies by height but roughly 100MB
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else {
        dev_CompactedOutput = (int*)gpuMemoryBlock.getCompactedOutputGPUPointer();
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_p, noiseInput->getPArray(), 512 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_chances, chances, 16 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_regionArrayX, noiseInput->getRegionArrayX(), X_ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_regionArrayY, noiseInput->getRegionArrayY(), Y_ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_regionArrayZ, noiseInput->getRegionArrayZ(), height * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    std::clock_t c_end = std::clock();
    double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for inputs: %lf\n", time_elapsed_ms);







    // Launch a kernel on the GPU with one thread for each element.
    hipStream_t stream;
    hipStreamCreate(&stream);
    dim3 gridShape(X_ARRAY_SIZE, Y_ARRAY_SIZE, 1);
    generateNoise <<<gridShape, totalHeight,0, stream >>>(dev_p,dev_chances, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output,totalHeight);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for computation before sleep: %lf\n", time_elapsed_ms);
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    while (hipStreamQuery(stream) == hipErrorNotReady) {
        Sleep(10);
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING)printf("Clocktime for computation before synchronize: %lf\n", time_elapsed_ms);

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for computation after synchronize: %lf\n", time_elapsed_ms);

    thrust::device_ptr<bool> t_output(dev_output);
    thrust::device_ptr<int> t_compactedOutput(dev_CompactedOutput);
    thrust::device_vector<bool> d_outputVector(t_output, t_output + CURRENT_OUTPUT_SIZE);
    thrust::device_vector<int> d_compactedOutputVector(t_compactedOutput, t_compactedOutput + CURRENT_OUTPUT_SIZE);

    thrust::device_vector<int>::iterator t_compactedOutputEnd =
        thrust::copy_if(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(CURRENT_OUTPUT_SIZE), d_outputVector.begin(), d_compactedOutputVector.begin(), is_true());

    outputSize = thrust::distance(d_compactedOutputVector.begin(), t_compactedOutputEnd);

    output = new int[outputSize];

    thrust::copy(d_compactedOutputVector.begin(), t_compactedOutputEnd, output);

    //save pointers for reuse
    gpuMemoryBlock.setpGPUPointer((long long)dev_p);
    gpuMemoryBlock.setxGPUPointer((long long)dev_regionArrayX);
    gpuMemoryBlock.setyGPUPointer((long long)dev_regionArrayY);
    gpuMemoryBlock.setzGPUPointer((long long)dev_regionArrayZ);
    gpuMemoryBlock.setOutputGPUPointer((long long)dev_output);
    gpuMemoryBlock.setCompactedOutputGPUPointer((long long)dev_CompactedOutput);

    hipFree(dev_chances);

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for after copying results: %lf\n", time_elapsed_ms);
    
    return cudaStatus;
}



void freeCudaMemory(int* dev_p, float* dev_regionArrayX, float* dev_regionArrayY, float* dev_regionArrayZ, bool* dev_output, int* dev_compactedOutput) {
    hipFree(dev_regionArrayX);
    hipFree(dev_regionArrayY);
    hipFree(dev_regionArrayZ);
    hipFree(dev_output);
    hipFree(dev_compactedOutput);
}

jobject createResponse(JNIEnv* env, int size, int* output, GPUMemoryBlock gpuMemoryBlock) {
    jclass noiseHardwareAcceleratorResponseClass = env->FindClass("org/pepsoft/worldpainter/exporting/NoiseHardwareAcceleratorResponse");
    jclass gpuMemoryBlockClass = env->FindClass("org/pepsoft/worldpainter/exporting/gpuacceleration/GPUMemoryBlock");

    jmethodID responseConstructorMethod = env->GetMethodID(noiseHardwareAcceleratorResponseClass, "<init>", "([ILorg/pepsoft/worldpainter/exporting/gpuacceleration/GPUMemoryBlock;)V");

    jintArray result = env->NewIntArray(size);
    env->SetIntArrayRegion(result, 0, size, (jint*)output);

    delete[] output;

    jobject gpuMemoryBlockObject = gpuMemoryBlock.getJavaObject();
    jobject response = env->NewObject(noiseHardwareAcceleratorResponseClass, responseConstructorMethod, result, gpuMemoryBlockObject);

    return response;
}

JNIEXPORT jobject JNICALL Java_org_pepsoft_worldpainter_exporting_gpuacceleration_ResourceNoiseGenerationRequest_getResourceRegionNoiseData(JNIEnv* env, jobject obj, jobject gpuNoiseRequestObject) {
    std::clock_t c_start = std::clock();

    GPUNoiseRequest gpuNoiseRequest = GPUNoiseRequest(env, gpuNoiseRequestObject);
    ResourceNoiseGenerationRequest resourceNoiseGenerationRequest = gpuNoiseRequest.getResourcesNoiseGenerationRequest();
    GPUMemoryBlock gpuMemoryBlock = gpuNoiseRequest.getGPUMemoryBlock();

    int outputSize;
    int* outputArray;


    const int totalHeight = resourceNoiseGenerationRequest.getMaxHeight() - resourceNoiseGenerationRequest.getMinHeight();

    JavaRandom random = JavaRandom(env, resourceNoiseGenerationRequest.getSeed());

    NoiseInput*  resourceNoiseInput =new NoiseInput(resourceNoiseGenerationRequest, random);

    hipError_t cudaStatus = noiseWithCuda(resourceNoiseGenerationRequest.getChances(),resourceNoiseInput, outputArray, outputSize, totalHeight, gpuMemoryBlock);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return NULL;
    }

    jobject result = createResponse(env,outputSize, outputArray, gpuMemoryBlock);

    std::clock_t c_end = std::clock();
    double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Finished in: %f Clocktime: %lf\n\n", time_elapsed_ms);
    
    return result;
}
