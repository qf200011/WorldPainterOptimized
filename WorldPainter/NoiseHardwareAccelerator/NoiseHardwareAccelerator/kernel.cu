#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "org_pepsoft_worldpainter_exporting_gpuacceleration_ResourceNoiseGenerationRequest.h"
#include <stdlib.h>
#include "NoiseGeneration.h"
#include <windows.h>
#include <jni.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include <chrono>
#include <stdio.h>
#include <ctime>

#define TILE_SIZE 128
#define REGION_SIZE 512
#define CHUNK_SIZE 16
#define TILES_PER_REGION_AXIS 4
#define CHUNKS_PER_TILE_AXIS 8
#define MAX_HEIGHT 128
#define MIN_HEIGHT -64
#define X_ARRAY_SIZE 512
#define Y_ARRAY_SIZE 512
#define DEBUGGING false
#define height 32

hipError_t noiseWithCuda(int* p, float* chances, float* regionArrayX, float* regionArrayY, float* regionArrayZ, bool* output, int& outputSize, int totalHeight, long long& dev_regionArrayXPtr, long long& dev_regionArrayYPtr, long long& dev_regionArrayZPtr, long long& dev_pPtr, long long& dev_outputPtr, long long& dev_compactedOutputPtr);
void getRegionArray(float* regionArrayX, float* regionArrayY, float* regionArrayZ, int minHeight, int maxHeight, int heightOffset, int regionX, int regionY, float blobSize);
void getPArray(int* p, JNIEnv* env, jlong seed);
void swap(int* array, int index1, int index2);
void freeCudaMemory(int* dev_p, float* dev_regionArrayX, float* dev_regionArrayY, float* dev_regionArrayZ, bool* dev_output, int* dev_compactedOutput);


__global__ void generateNoise(int* p, float* chances, float* regionArrayX, float* regionArrayY, float* regionArrayZ, bool *output, int totalHeight)
{
    float outputNoise = getPerlinNoiseAt(regionArrayX[blockIdx.x], regionArrayY[blockIdx.y], regionArrayZ[threadIdx.x], p);

    bool shouldSetMaterial = outputNoise >= chances[8];

    output[blockIdx.x + (blockIdx.y * X_ARRAY_SIZE) + (threadIdx.x * X_ARRAY_SIZE * X_ARRAY_SIZE)] = shouldSetMaterial;

}

struct is_true {
    __host__ __device__
        bool operator() (const bool success) {
        return success;
    }
};



// Helper function for using CUDA to add vectors in parallel.
hipError_t noiseWithCuda(int* p, float* chances, float* regionArrayX, float* regionArrayY, float* regionArrayZ, int*&  output,int& outputSize, int totalHeight, long long &dev_regionArrayXPtr, long long &dev_regionArrayYPtr, long long& dev_regionArrayZPtr, long long& dev_pPtr, long long &dev_outputPtr, long long& dev_compactedOutputPtr)
{
    std::clock_t c_start = std::clock();

    int* dev_p;
    float* dev_regionArrayX;
    float* dev_regionArrayY;
    float* dev_regionArrayZ;
    bool* dev_output;
    int* dev_CompactedOutput;
    float* dev_chances;

    const int MAX_OUTPUT_SIZE = X_ARRAY_SIZE * Y_ARRAY_SIZE * height;
    const int CURRENT_OUTPUT_SIZE = X_ARRAY_SIZE * Y_ARRAY_SIZE * totalHeight;


    hipError_t cudaStatus;
    auto t_start = std::chrono::high_resolution_clock::now();
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    if (dev_pPtr == 0){
        cudaStatus = hipMalloc((void**)&dev_p, 512 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_p = (int*)dev_pPtr;
    }

    cudaStatus = hipMalloc((void**)&dev_chances, 16 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    if (dev_regionArrayXPtr==0){
    cudaStatus = hipMalloc((void**)&dev_regionArrayX, X_ARRAY_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        fprintf(stderr, hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }
    }
    else { //reuse
        dev_regionArrayX = (float*)dev_regionArrayXPtr;
    }

    if (dev_regionArrayYPtr == 0) {
        cudaStatus = hipMalloc((void**)&dev_regionArrayY, Y_ARRAY_SIZE * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_regionArrayY = (float*)dev_regionArrayYPtr;
    }

    if (dev_regionArrayZPtr == 0){
        cudaStatus = hipMalloc((void**)&dev_regionArrayZ, height * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_regionArrayZ = (float*)dev_regionArrayZPtr;
    }

    if (dev_outputPtr==0)
    {
        cudaStatus = hipMalloc((void**)&dev_output,sizeof(bool)* MAX_OUTPUT_SIZE); //Varies by height but roughly 25MB
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else { //reuse
        dev_output = (bool*)dev_outputPtr;
    }
    if (dev_compactedOutputPtr == 0) {
        cudaStatus = hipMalloc((void**)&dev_CompactedOutput, (MAX_OUTPUT_SIZE) * sizeof(int)); // Varies by height but roughly 100MB
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
            return cudaStatus;
        }
    }
    else {
        dev_CompactedOutput = (int*)dev_compactedOutputPtr;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_p, p, 512 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_chances, chances, 16 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_regionArrayX, regionArrayX, X_ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_regionArrayY, regionArrayY, Y_ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_regionArrayZ, regionArrayZ, height * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    std::clock_t c_end = std::clock();
    double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for inputs: %lf\n", time_elapsed_ms);







    // Launch a kernel on the GPU with one thread for each element.
    hipStream_t stream;
    hipStreamCreate(&stream);
    dim3 gridShape(X_ARRAY_SIZE, Y_ARRAY_SIZE, 1);
    generateNoise <<<gridShape, totalHeight,0, stream >>>(dev_p,dev_chances, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output,totalHeight);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        freeCudaMemory(dev_p, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output, dev_CompactedOutput);
        return cudaStatus;
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for computation before sleep: %lf\n", time_elapsed_ms);
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    while (hipStreamQuery(stream) == hipErrorNotReady) {
        Sleep(10);
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING)printf("Clocktime for computation before synchronize: %lf\n", time_elapsed_ms);

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for computation after synchronize: %lf\n", time_elapsed_ms);

    thrust::device_ptr<bool> t_output(dev_output);
    thrust::device_ptr<int> t_compactedOutput(dev_CompactedOutput);
    thrust::device_vector<bool> d_outputVector(t_output, t_output + CURRENT_OUTPUT_SIZE);
    thrust::device_vector<int> d_compactedOutputVector(t_compactedOutput, t_compactedOutput + CURRENT_OUTPUT_SIZE);

    thrust::device_vector<int>::iterator t_compactedOutputEnd =
        thrust::copy_if(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(CURRENT_OUTPUT_SIZE), d_outputVector.begin(), d_compactedOutputVector.begin(), is_true());

    outputSize = thrust::distance(d_compactedOutputVector.begin(), t_compactedOutputEnd);

    output = new int[outputSize];

    thrust::copy(d_compactedOutputVector.begin(), t_compactedOutputEnd, output);

    //save pointers for reuse
    dev_pPtr = (long long)dev_p;
    dev_regionArrayXPtr = (long long)dev_regionArrayX;
    dev_regionArrayYPtr = (long long)dev_regionArrayY;
    dev_regionArrayZPtr = (long long)dev_regionArrayZ;
    dev_outputPtr = (long long)dev_output;
    dev_compactedOutputPtr = (long long)dev_CompactedOutput;

    hipFree(dev_chances);

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Clocktime for after copying results: %lf\n", time_elapsed_ms);
    
    return cudaStatus;
}



void freeCudaMemory(int* dev_p, float* dev_regionArrayX, float* dev_regionArrayY, float* dev_regionArrayZ, bool* dev_output, int* dev_compactedOutput) {
    hipFree(dev_regionArrayX);
    hipFree(dev_regionArrayY);
    hipFree(dev_regionArrayZ);
    hipFree(dev_output);
    hipFree(dev_compactedOutput);
}


void getRegionArray(float* regionArrayX, float* regionArrayY, float* regionArrayZ, int minHeight, int maxHeight, int regionX, int regionY, float blobSize, int heightOffset) {

    int minTileX = regionX * 4;
    int minTileY = regionY * 4;

    for (int tileX = 0; tileX < TILES_PER_REGION_AXIS; tileX++) {
        for (int x = 0; x < CHUNK_SIZE * CHUNKS_PER_TILE_AXIS; x++) {
            int worldX = (tileX+minTileX)*TILE_SIZE + x;
            regionArrayX[tileX * TILE_SIZE + x] = worldX / blobSize;
        }
    }

    for (int tileY = 0; tileY < TILES_PER_REGION_AXIS; tileY++) {
        for (int y = 0; y < CHUNK_SIZE * CHUNKS_PER_TILE_AXIS; y++) {
            int worldY = (tileY+minTileY)*TILE_SIZE + y;
            regionArrayY[tileY * TILE_SIZE + y] = worldY / blobSize;
        }
    }
    int totalHeight = maxHeight - minHeight;

    for (int z = 0; z < totalHeight; z++) {
        regionArrayZ[z] = (z+minHeight) / blobSize;
    }
}

void getPArray(int* p, JNIEnv* env,jlong seed) {
    int permutation[256];

    for (int i=0; i < 256; i++) {
        permutation[i] = i;
    }

    jclass randomClass = env->FindClass("java/util/Random"); //use Java so we can keep the same seed.
    if (randomClass == NULL) {
        fprintf(stderr, "Unable to find java Random object");
        return;
    }
    jmethodID randomConstructor = env->GetMethodID(randomClass, "<init>", "(J)V");
    if (randomConstructor == NULL) {
        fprintf(stderr, "Unable to find java Random object constructor");
        return;
    }

    jmethodID nextIntMethod = env->GetMethodID(randomClass, "nextInt", "(I)I");

    jobject randomObject = env->NewObject(randomClass, randomConstructor, seed);


    for (int i = 256; i > 1; i--) {
        jint randomInt = env->CallIntMethod(randomObject, nextIntMethod, i); //random.NextInt(i)
        swap(permutation, i-1, randomInt);
    }

    for (int i = 0; i < 256; i++) {
        p[256 + i] = p[i] = permutation[i];
    }


}

void swap(int* array, int index1, int index2) {
    int temp = array[index1];
    array[index1] = array[index2];
    array[index2] = temp;
}

void getDataFromRequest(JNIEnv* env, jobject request, jlong& materialSeed, jint& regionX, jint& regionY, jint& materialMinHeight, jint& materialMaxHeight, jint& heightOffset, jfloat& blobSize, jlong& regionXPtr, jlong& regionYPtr, jlong& regionZPtr, jlong& pPtr, jlong& outputPtr, jlong& compactedOutputPtr, int*& outputArray, float*& chances) {
    jclass gpuNoiseRequestClass = env->FindClass("org/pepsoft/worldpainter/exporting/NoiseHardwareAccelerator$GPUNoiseRequest");
    jclass gpuMemoryBlockClass = env->FindClass("org/pepsoft/worldpainter/exporting/gpuacceleration/GPUMemoryBlock");
    
    jclass noiseHardwareAcceleratorRequestClass = env->FindClass("org/pepsoft/worldpainter/exporting/gpuacceleration/ResourceNoiseGenerationRequest");

    jmethodID getNoiseRequestMethod = env->GetMethodID(gpuNoiseRequestClass, "getNoiseGenerationRequest", "()Lorg/pepsoft/worldpainter/exporting/gpuacceleration/NoiseGenerationRequest;");
    jobject noiseGnerationRequest = env->CallObjectMethod(request, getNoiseRequestMethod);

    jmethodID getMaterialSeedMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getSeed", "()J");
    jmethodID getRegionXMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getRegionX", "()I");
    jmethodID getRegionYMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getRegionY", "()I");
    jmethodID getMaterialMinHeightMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getMinHeight", "()I");
    jmethodID getMaterialMaxHeightMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getMaxHeight", "()I");
    jmethodID getMaterialHeightOffset = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getHeightOffset", "()I");
    jmethodID getBlobSizeMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getBlobSize", "()F");
    jmethodID getChancesMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getChances", "()[F");

    jmethodID getGPUMemoryBlockMethodId = env->GetMethodID(gpuNoiseRequestClass, "getGpuMemoryBlock", "()Lorg/pepsoft/worldpainter/exporting/gpuacceleration/GPUMemoryBlock;");
    jobject gpuMemoryBlock = env->CallObjectMethod(request, getGPUMemoryBlockMethodId);

    jmethodID getxGPUPointerMethod = env->GetMethodID(gpuMemoryBlockClass, "getxGPUPointer", "()J");
    jmethodID getyGPUPointerMethod = env->GetMethodID(gpuMemoryBlockClass, "getyGPUPointer", "()J");
    jmethodID getzGPUPointerMethod = env->GetMethodID(gpuMemoryBlockClass, "getzGPUPointer", "()J");
    jmethodID getpGPUPointerMethod = env->GetMethodID(gpuMemoryBlockClass, "getpGPUPointer", "()J");
    jmethodID getOutputGPUPointerMethod = env->GetMethodID(gpuMemoryBlockClass, "getOutputGPUPointer", "()J");
    jmethodID getCompactedGPUPointerMethod = env->GetMethodID(gpuMemoryBlockClass, "getCompactedGPUPointer", "()J");

    materialSeed = env->CallLongMethod(noiseGnerationRequest, getMaterialSeedMethod);
    regionX = env->CallIntMethod(noiseGnerationRequest, getRegionXMethod);
    regionY = env->CallIntMethod(noiseGnerationRequest, getRegionYMethod);
    materialMinHeight = env->CallIntMethod(noiseGnerationRequest, getMaterialMinHeightMethod);
    materialMaxHeight = env->CallIntMethod(noiseGnerationRequest, getMaterialMaxHeightMethod);
    heightOffset = env->CallIntMethod(noiseGnerationRequest, getMaterialHeightOffset);
    blobSize = env->CallFloatMethod(noiseGnerationRequest, getBlobSizeMethod);
    regionXPtr = env->CallLongMethod(gpuMemoryBlock, getxGPUPointerMethod);
    regionYPtr = env->CallLongMethod(gpuMemoryBlock, getyGPUPointerMethod);
    regionZPtr = env->CallLongMethod(gpuMemoryBlock, getzGPUPointerMethod);
    pPtr = env->CallLongMethod(gpuMemoryBlock, getpGPUPointerMethod);
    outputPtr = env->CallLongMethod(gpuMemoryBlock, getOutputGPUPointerMethod);
    compactedOutputPtr = env->CallLongMethod(gpuMemoryBlock, getCompactedGPUPointerMethod);
    jfloatArray chancesArray = (jfloatArray)env->CallObjectMethod(noiseGnerationRequest, getChancesMethod);
    chances = env->GetFloatArrayElements(chancesArray, 0);
}


jobject createResponse(JNIEnv* env, int size, int* output, long long dev_regionXPtr, long long dev_regionYPtr, long long dev_regionZPtr, long long dev_pPtr, long long dev_outputPtr, long long dev_compactedOutputPtr, int totalHeight) {
    jclass noiseHardwareAcceleratorResponseClass = env->FindClass("org/pepsoft/worldpainter/exporting/NoiseHardwareAcceleratorResponse");
    jclass gpuMemoryBlockClass = env->FindClass("org/pepsoft/worldpainter/exporting/gpuacceleration/GPUMemoryBlock");

    jmethodID responseConstructorMethod = env->GetMethodID(noiseHardwareAcceleratorResponseClass, "<init>", "([ILorg/pepsoft/worldpainter/exporting/gpuacceleration/GPUMemoryBlock;)V");
    jmethodID gpuMemoryConstructorMethod = env->GetMethodID(gpuMemoryBlockClass, "<init>", "(JJJJJJ)V");

    jintArray result = env->NewIntArray(size);
    env->SetIntArrayRegion(result, 0, size, (jint*)output);

    delete[] output;

    jlong pPtr = (jlong)dev_pPtr;
    jlong regionXPtr = (jlong)dev_regionXPtr;
    jlong regionYPtr = (jlong)dev_regionYPtr;
    jlong regionZPtr = (jlong)dev_regionZPtr;
    jlong outputPtr = (jlong)dev_outputPtr;
    jlong compactedOutputPtr = (jlong)dev_compactedOutputPtr;

    jobject gpuMemoryBlock = env->NewObject(gpuMemoryBlockClass, gpuMemoryConstructorMethod, regionXPtr, regionYPtr, regionZPtr, pPtr, outputPtr,compactedOutputPtr);
    jobject response = env->NewObject(noiseHardwareAcceleratorResponseClass, responseConstructorMethod, result, gpuMemoryBlock);

    return response;
}

JNIEXPORT jobject JNICALL Java_org_pepsoft_worldpainter_exporting_gpuacceleration_ResourceNoiseGenerationRequest_getResourceRegionNoiseData(JNIEnv* env, jobject obj, jobject request) {
    std::clock_t c_start = std::clock();

    jlong materialSeed;
    jint regionX;
    jint regionY;
    jint materialMinHeight;
    jint materialMaxHeight;
    jint heightOffset;
    jlong dev_regionXPtr;
    jlong dev_regionYPtr;
    jlong dev_regionZPtr;
    jlong dev_pPtr;
    jlong dev_outputPtr;
    jlong dev_compactedOutputPtr;
    int* outputArray;
    float* chances;
    int outputSize;
    float blobSize;

    getDataFromRequest(env, request, materialSeed, regionX, regionY, materialMinHeight, materialMaxHeight,heightOffset, blobSize, dev_regionXPtr, dev_regionYPtr, dev_regionZPtr, dev_pPtr, dev_outputPtr, dev_compactedOutputPtr, outputArray, chances);


    const int totalHeight = materialMaxHeight - materialMinHeight;

    float regionArrayX[X_ARRAY_SIZE]; //dx
    float regionArrayY[Y_ARRAY_SIZE]; //dy
    float* regionArrayZ; //dz but shifted
    int p[512];
    //[TILE_SIZE * TILES_PER_REGION_AXIS] [TILE_SIZE * TILES_PER_REGION_AXIS] [totalHeight]
    regionArrayZ = new float[totalHeight];

    getRegionArray(regionArrayX, regionArrayY, regionArrayZ, materialMinHeight, materialMaxHeight, regionX, regionY,blobSize,heightOffset);


    getPArray(p, env, materialSeed);
    //double test = getPerlinNoiseAt(regionArrayX[128], regionArrayY[128], regionArrayZ[60 + 64],p);


    // Add vectors in parallel.

    hipError_t cudaStatus = noiseWithCuda(p, chances, regionArrayX, regionArrayY, regionArrayZ, outputArray, outputSize, totalHeight, dev_regionXPtr, dev_regionYPtr, dev_regionZPtr, dev_pPtr, dev_outputPtr, dev_compactedOutputPtr);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        delete[] regionArrayZ;
        return NULL;
    }

    jobject result = createResponse(env,outputSize, outputArray, dev_regionXPtr, dev_regionYPtr, dev_regionZPtr, dev_pPtr,dev_outputPtr, dev_compactedOutputPtr, totalHeight);
   
    delete[] regionArrayZ;

    std::clock_t c_end = std::clock();
    double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    if (DEBUGGING) printf("Finished in: %f Clocktime: %lf\n\n", time_elapsed_ms);
    
    return result;
}
